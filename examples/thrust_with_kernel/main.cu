#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <iostream>
#include <stdexcept>

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/generate.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>

const size_t SIZE = 1e6; // 1,000,000



struct RandomFloatGenerator
{
	RandomFloatGenerator()
	{
		srand(time(0));
	}

	float operator()() const
	{
		return static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
	}
};


float *cpu_add_vec(float *arr1, float *arr2, size_t size1, size_t size2)
{
	if(size1 != size2)
	{
		throw std::runtime_error("Tried to cpu add vec when sizes are not the same!");
	}

	float *output_arr = new float[size1];

	for(size_t i = 0; i < size1; i++)
	{
		output_arr[i] = arr1[i] + arr2[i];
	}

	return output_arr;
}


float cpu_median(float *arr, size_t size)
{
	size_t midpoint = size / 2;

	std::nth_element(arr, arr + midpoint, arr + size);

	if(size & 0b01 == 0)
	{
		float mid1 = arr[midpoint];
		std::nth_element(arr, arr + midpoint - 1, arr + size);
		float mid2 = arr[midpoint - 1];
		return (mid1 + mid2) / 2.0f;
	}

	return arr[midpoint];
}

float *generate_array(size_t size)
{
	float *data = new float[size];

	for(size_t i = 0; i < size; i++)
	{
		float r = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
		data[i] = r;
	}

	return data;
}


// CUDA vector addition, this happens before using thrust
__global__ void vector_add_kernel(const float *a, const float *b, float *c, size_t size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < size)
	{
		c[idx] = a[idx] + b[idx];
	}
}

__host__ float thrust_find_median(float *device_data, size_t size)
{
	thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(device_data);

	// This sorts the device ptr in place
	thrust::sort(thrust::device, dev_ptr, dev_ptr + size);

	size_t midpoint = size / 2;
	float  mid1     = dev_ptr[midpoint];

	if(size & 0b01 == 0)
	{
		float mid2 = dev_ptr[midpoint - 1];
		return (mid1 + mid2) / 2.0f;
	}

	return mid1;
}



int main()
{
	srand(time(0));

	// Time data creation
	std::chrono::high_resolution_clock::time_point start_time = std::chrono::high_resolution_clock::now();

	float *data1 = generate_array(SIZE);
	float *data2 = generate_array(SIZE);

	std::chrono::high_resolution_clock::time_point end_time = std::chrono::high_resolution_clock::now();

	std::chrono::milliseconds dt_ms = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);
	std::chrono::microseconds dt_us = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);

	printf("CPU Time to generate data: %lu ms %lu us\n", dt_ms.count(), dt_us.count());


	// Time CPU add
	start_time = std::chrono::high_resolution_clock::now();

	float *summed_vecs = cpu_add_vec(data1, data2, SIZE, SIZE);

	end_time = std::chrono::high_resolution_clock::now();
	dt_ms    = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);
	dt_us    = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);

	printf("CPU add time: %lu ms %lu us\n", dt_ms.count(), dt_us.count());


	// Time finding median
	start_time = std::chrono::high_resolution_clock::now();

	float median = cpu_median(summed_vecs, SIZE);

	end_time = std::chrono::high_resolution_clock::now();
	dt_ms    = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);
	dt_us    = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);

	printf("CPU Median: %f\nTime to find CPU median: %lu ms %lu us\n", median, dt_ms.count(), dt_us.count());

	// Cleanup sooner rather than later
	delete[] data1;
	delete[] data2;
	delete[] summed_vecs;

	// Generate 1M random numbers on host
	start_time = std::chrono::high_resolution_clock::now();

	float *data_a;
	float *data_b;
	float *data_c; // output
	hipMalloc(&data_a, SIZE * sizeof(float));
	hipMalloc(&data_b, SIZE * sizeof(float));
	hipMalloc(&data_c, SIZE * sizeof(float));

	thrust::host_vector<float> host_vec_a(SIZE);
	thrust::host_vector<float> host_vec_b(SIZE);

	thrust::generate(host_vec_a.begin(), host_vec_a.end(), RandomFloatGenerator());
	thrust::generate(host_vec_b.begin(), host_vec_b.end(), RandomFloatGenerator());


	end_time = std::chrono::high_resolution_clock::now();
	dt_ms    = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);
	dt_us    = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);
	printf("Thrust Time to generate data: %lu ms %lu us\n", dt_ms.count(), dt_us.count());


	// Transfer to device
	start_time = std::chrono::high_resolution_clock::now();

	hipMemcpy(data_a, host_vec_a.data(), SIZE * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(data_b, host_vec_b.data(), SIZE * sizeof(float), hipMemcpyHostToDevice);

	end_time = std::chrono::high_resolution_clock::now();
	dt_ms    = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);
	dt_us    = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);

	printf("CUDA data transfer time (for generated data, coming from CPU generation): %lu ms %lu us\n", dt_ms.count(), dt_us.count());

	// Time vector add kernel


	start_time = std::chrono::high_resolution_clock::now();

	int threadcount = 256;
	int blocks      = (SIZE + threadcount - 1) / threadcount;
	vector_add_kernel<<<blocks, threadcount>>>(data_a, data_b, data_c, SIZE);
	hipDeviceSynchronize();

	end_time = std::chrono::high_resolution_clock::now();
	dt_ms    = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);
	dt_us    = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);

	printf("CUDA vector add kernel time: %lu ms %lu ms\n", dt_ms.count(), dt_us.count());

	// Find median with thrust
	start_time = std::chrono::high_resolution_clock::now();

	float gpu_median = thrust_find_median(data_c, SIZE);

	end_time = std::chrono::high_resolution_clock::now();
	dt_ms    = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);
	dt_us    = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time);

	printf("Thrust Median %f\nThrust Time to find median: %lu ms %lu us\n", gpu_median, dt_ms.count(), dt_us.count());

	return 0;
}
